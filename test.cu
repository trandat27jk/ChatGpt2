
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to add two arrays element-wise
__global__
void addArrays(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int n = 100000;  // Number of elements in arrays
    float *h_a, *h_b, *h_c;  // Host arrays
    float *d_a, *d_b, *d_c;  // Device arrays
    size_t size = n * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize arrays
    for (int i = 0; i < n; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch kernel
    addArrays<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify results (printing first 10 elements)
    std::cout << "Results (first 10 elements):" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
